#include "hip/hip_runtime.h"
#include <vector>

#include "caffe/layers/conv_layer.hpp"

namespace caffe {

vector <float *> pruning1;
vector <float *> pruning2;
vector <float *> pruning3;
int epoch=0;
int images=0;

template <typename Dtype>
__global__ void Threshold_pruning(const int n,
    Dtype* in, float* out) {
  CUDA_KERNEL_LOOP(index, n) {
	  if (in[index]>0.5)
		  out[index] =1234;
	  else
		  out[index] = in[index];
  }
}

template <typename Dtype>
void ConvolutionLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
      const vector<Blob<Dtype>*>& top) {
	const int p_t =0; 
	const int Z=1;
  const Dtype* weight = this->blobs_[0]->gpu_data();
  for (int i = 0; i < bottom.size(); ++i) {
    const Dtype* bottom_data = bottom[i]->gpu_data();
    Dtype* top_data = top[i]->mutable_gpu_data();
    const int count = this->top_dim_;
    if (epoch==0 && this->phase_ == TRAIN){
	    float* d_x = NULL;
	    hipMalloc(&d_x, sizeof(float)*count*this->num_);
	    hipMemset(d_x,0,count*this->num_*sizeof(float));
	    if (count == 32768)//MNIST: 11520,3200
	         pruning1.push_back(d_x);
	    else if (count==8192)
		 pruning2.push_back(d_x);
	    else
        	 pruning3.push_back(d_x);
    }
    for (int n = 0; n < this->num_; ++n) {
      if (this->phase_ == TEST || ((this->phase_ == TRAIN) && (epoch<p_t))){
          this->forward_gpu_gemm(bottom[i]->count() / this->num_,bottom_data + n * this->bottom_dim_, this->blobs_[0]->count(), weight,
              top[i]->count() / this->num_, top_data + n * this->top_dim_, NULL,true);
      } else {
	  if ((epoch-p_t)% (Z+1)==0){
		this->forward_gpu_gemm(bottom[i]->count() / this->num_, bottom_data + n * this->bottom_dim_, this->blobs_[0]->count(), weight,
	              top[i]->count() / this->num_, top_data + n * this->top_dim_,NULL,true);
		if (count == 32768)
			 Threshold_pruning<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(count, top_data + n * this->top_dim_, pruning1[images/100]+n * this->top_dim_);
		else if (count==8192)
			 Threshold_pruning<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(count, top_data + n * this->top_dim_, pruning2[images/100]+n * this->top_dim_);
		else 
			 Threshold_pruning<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(count, top_data + n * this->top_dim_, pruning3[images/100]+n * this->top_dim_);
		CUDA_POST_KERNEL_CHECK;
	  } else {
		if (count == 32768)
			this->forward_gpu_gemm(bottom[i]->count() / this->num_, bottom_data + n * this->bottom_dim_, this->blobs_[0]->count(), weight, top[i]->count() / this->num_, top_data + n * this->top_dim_, pruning1[images/100]+n * this->top_dim_, false);
		else if (count==8192)
			this->forward_gpu_gemm(bottom[i]->count() / this->num_, bottom_data + n * this->bottom_dim_, this->blobs_[0]->count(), weight, top[i]->count() / this->num_, top_data + n * this->top_dim_, pruning2[images/100]+n * this->top_dim_, false);
		else 
			this->forward_gpu_gemm(bottom[i]->count() / this->num_, bottom_data + n * this->bottom_dim_, this->blobs_[0]->count(), weight, top[i]->count() / this->num_, top_data + n * this->top_dim_, pruning3[images/100]+n * this->top_dim_, false); 
	  }
      }
      if (this->bias_term_) {
        const Dtype* bias = this->blobs_[1]->gpu_data();
      }
    }
    if (this->phase_ == TRAIN){
	if(this->top_dim_ == 4096)
		images+=this->num_;
	if (images >=50000){
		epoch++;
		images=0;
	}
    }
  }
}

template <typename Dtype>
void ConvolutionLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
      const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  const Dtype* weight = this->blobs_[0]->gpu_data();
  Dtype* weight_diff = this->blobs_[0]->mutable_gpu_diff();
  for (int i = 0; i < top.size(); ++i) {
    const Dtype* top_diff = top[i]->gpu_diff();
    // Bias gradient, if necessary.
    if (this->bias_term_ && this->param_propagate_down_[1]) {
      Dtype* bias_diff = this->blobs_[1]->mutable_gpu_diff();
      for (int n = 0; n < this->num_; ++n) {
        this->backward_gpu_bias(bias_diff, top_diff + n * this->top_dim_);
      }
    }
    if (this->param_propagate_down_[0] || propagate_down[i]) {
      const Dtype* bottom_data = bottom[i]->gpu_data();
      Dtype* bottom_diff = bottom[i]->mutable_gpu_diff();
      for (int n = 0; n < this->num_; ++n) {
        // gradient w.r.t. weight. Note that we will accumulate diffs.
        if (this->param_propagate_down_[0]) {
          this->weight_gpu_gemm(bottom_data + n * this->bottom_dim_,
              top_diff + n * this->top_dim_, weight_diff);
        }
        // gradient w.r.t. bottom data, if necessary.
        if (propagate_down[i]) {
          this->backward_gpu_gemm(top_diff + n * this->top_dim_, weight,
              bottom_diff + n * this->bottom_dim_);
        }
      }
    }
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(ConvolutionLayer);

}  // namespace caffe
